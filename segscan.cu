#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

#include <math.h>
#include <chrono>
#include <hip/hip_cooperative_groups.h>

#include "util.hpp"
#include "serial.hpp"

#define CUDA_CHECK(ans)                                                                  \
    {                                                                                    \
        gpuAssert((ans), __FILE__, __LINE__);                                            \
    }
   inline void
gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if(code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if(abort)
            exit(code);
    }
}


__global__ void naive_scan(float* x, float* y, size_t n) {
  size_t tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < n) {
    y[tid] =  x[tid];
  }

  __syncthreads();

  size_t bound = (size_t) (log2f(n) + 0.5);

  for (size_t d = 1; d <= bound; d++) {
    size_t d2 = powf(2, d-1);  
    if (tid >= d2) {
      size_t offset = (size_t) (powf(2, d-1)); 
      y[tid] = y[tid] + y[tid - offset];
    }
    __syncthreads();

  }
}


__global__ void prescan(float *g_idata, float *g_odata, int n) {
   int np2 = powf(2, ceil(log2f(n))); // next power-of-2 >= n

   extern __shared__ float temp[];
  
  int thid = blockIdx.x*blockDim.x + threadIdx.x;
  int offset = 1;

  __syncthreads();

  // load input into shared memory
  temp[2*thid] = (2*thid < n) ? g_idata[2*thid] : 0;
  temp[2*thid+1] = (2*thid+1 < n) ? g_idata[2*thid+1] : 0; 

  // build sum in place up the tree
  for (int d = np2>>1; d > 0; d >>= 1) {
    __syncthreads();
    if (thid < d)    { 
      int ai = offset*(2*thid+1)-1;  // k + 2^d - 1
      int bi = offset*(2*thid+2)-1;  // k + 2^d+1 - 1
      if (bi < np2 && ai < np2) {
	temp[bi] += temp[ai];
      }
    }
    offset *= 2;
  }
  __syncthreads();

  // clear the last element
  if (thid == 0) {
    temp[np2 - 1] = 0;  
  } 

  // traverse down tree, swapping elements to build scan
  for (int d = 1; d < np2; d *= 2)    {
    offset >>= 1;
    __syncthreads();

    if (thid < d) { 
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      if (ai < np2 && bi < np2) {
        float t = temp[ai];
        temp[ai] = temp[bi];
        temp[bi] += t;
      }
    }
  }
  __syncthreads();

  // write results to device memory
  if (2*thid<n) {
    g_odata[2*thid] = temp[2*thid] + g_idata[2*thid];
  }
  if (2*thid+1<n) {
    g_odata[2*thid+1] = temp[2*thid+1] + g_idata[2*thid+1];
  }
 

}

// Requires: All arrays are of size n, which must be a power of 2
// Effect: Sets g_odata to be the segmented scan of g_idata with the flags flag_orig
//         Does not modify g_idata, or flag_orig
__global__ void segscan(float *g_idata, float *g_odata,
                        float *flag_orig, float *flag_scan, float* sums, float* sums_flags, int n){
  int thid = threadIdx.x;
  size_t block_offset = blockIdx.x * blockDim.x * 2;
  g_idata += block_offset;
  g_odata += block_offset;
  flag_orig += block_offset;
  flag_scan += block_offset;
  int offset = 1;

  extern __shared__ float shmem[];
  float * temp = &shmem[0];
  float * flags = &shmem[n];
  float * flags_temp = &shmem[2*n];
  
  __syncthreads();

  // load input into shared memory
  temp[2*thid] = (2*thid < n) ? g_idata[2*thid] : 0;
  temp[2*thid+1] = (2*thid+1 < n) ? g_idata[2*thid+1] : 0;
  flags_temp[2*thid] = flags[2*thid] = (2*thid < n) ? flag_orig[2*thid] : 0;
  flags_temp[2*thid+1] = flags[2*thid+1] = (2*thid+1 < n) ? flag_orig[2*thid+1] : 0;

  // build sum in place up the tree
  for (int d = n>>1; d > 0; d >>= 1) {
    __syncthreads();
    if (thid < d)    { 
      int ai = offset*(2*thid+1)-1;  // k + 2^d - 1
      int bi = offset*(2*thid+2)-1;  // k + 2^(d+1) - 1
      if (bi < n && ai < n) {
        flags_temp[bi] = fmax(flags_temp[ai], flags_temp[bi]);
	if (flags[bi] == 0.0) {
	  temp[bi] += temp[ai];
	}
	flags[bi] = (flags[bi]==1.0 || flags[ai] == 1.0) ? 1.0: 0.0;
      }
    }
    offset *= 2;
  }
  
  // clear the last element
  if (thid == 0) {
    temp[n - 1] = 0;
    flags_temp[n - 1] = 0;
  } 

  // traverse down tree & build scan
  for (int d = 1; d < n; d *= 2)    {
    offset >>= 1;
    __syncthreads();
    if (thid < d) { 
      int ai = offset*(2*thid+1)-1;  // k + 2^d  - 1
      int bi = offset*(2*thid+2)-1;  // k + 2^d+1 - 1
      float t = temp[ai];
      temp[ai] = temp[bi];
      if (ai < n && bi < n) {
        float tf = flags_temp[ai];
        flags_temp[ai] = flags_temp[bi];
        flags_temp[bi] = fmax(tf, flags_temp[bi]);
      }
      if (flag_orig[ai+1] == 1.0) {
	temp[bi] = 0;
      } else if (flags[ai] == 1.0) {
	temp[bi] = t;
      } else {
	temp[bi] += t;
      }
      flags[ai] = 0;
    }
  }

  __syncthreads();
  
  // write results to device memory
  if (2*thid < n) {
    g_odata[2*thid] = temp[2*thid] + g_idata[2*thid];
    flag_scan[2*thid] = fmax(flags_temp[2*thid], flag_orig[2*thid]);
  }
  if (2*thid+1 < n) {
    g_odata[2*thid+1] = temp[2*thid+1] + g_idata[2*thid+1];
    flag_scan[2*thid+1] = fmax(flags_temp[2*thid+1], flag_orig[2*thid+1]);
  }

  __syncthreads();

  if (thid == 0) {
    if (sums != nullptr)
      sums[blockIdx.x] = g_odata[blockDim.x * 2 - 1];
    if (sums_flags != nullptr)
      sums_flags[blockIdx.x] = flag_scan[blockDim.x * 2 - 1];
  }
}

__global__ void add_offsets(float *x, float* flags, float* sums, int n){
  size_t tid = threadIdx.x + blockIdx.x*blockDim.x;

  if (tid < n) {
    // TODO: add the `chunk_id-1`'th element of `sums`
    //       to every element, according to its `chunk_id`.
    x[tid] += (blockIdx.x / 2 > 0 ? sums[blockIdx.x / 2 - 1] : 0) * (1 - flags[tid]);
  }
}


float run_scan(std::vector<float> inv, std::vector<float> outv) {
  return 0.0;
}

float* run_segscan(float* d_x, float* d_f, size_t n) {
  int np2 = powf(2, ceil(log2f(n)));
  size_t blocksize = std::min<size_t>(np2/2, 1024);
  size_t nblocks = (np2/2) / blocksize;
  float *d_y, *d_fs;
  CUDA_CHECK(hipMalloc(&d_y, sizeof(float)*np2));
  CUDA_CHECK(hipMalloc(&d_fs, sizeof(float)*np2));
  if (nblocks == 1) {
    segscan<<<nblocks, blocksize, 6*blocksize*sizeof(float)>>>(d_x, d_y, d_f, d_fs, nullptr, nullptr, 2*blocksize);
    hipFree(d_fs);
    return d_y;
  }
  float *d_sums, *d_sums_f;
  CUDA_CHECK(hipMalloc(&d_sums, sizeof(float)*nblocks));
  CUDA_CHECK(hipMalloc(&d_sums_f, sizeof(float)*nblocks));
  segscan<<<nblocks, blocksize, 6*blocksize*sizeof(float)>>>(d_x, d_y, d_f, d_fs, d_sums, d_sums_f, 2*blocksize);
  float *d_sumscan = run_segscan(d_sums, d_sums_f, nblocks);
  hipFree(d_sums);
  hipFree(d_sums_f);
  add_offsets<<<(np2+blocksize-1)/blocksize, blocksize>>>(d_y, d_fs, d_sumscan, np2);
  hipFree(d_fs);
  hipFree(d_sumscan);
  return d_y;
}

int main(int argc, char** argv) {

  if (argc < 2) {
    fprintf(stderr, "scan [size of array]\n");
    return 1;
  }
  
  size_t n = std::atoi(argv[1]);
  int np2 = powf(2, ceil(log2f(n))); // next power-of-2 >= n
  
//  assert(n <= 2048);

  // 1 thread for every 2 elements (round up)
  size_t blocksize = std::min<size_t>(np2/2, 1024);
  size_t nblocks = (np2/2) / blocksize;

  std::vector<float> x = gen_vec(n);  // input
 
  auto begin = std::chrono::high_resolution_clock::now();
  std::vector<float> rv_cpu = cpu_scan(x);
  auto end = std::chrono::high_resolution_clock::now();
  double duration_cpu = std::chrono::duration<double>(end - begin).count();

  float* d_x;
  float* d_y;

  CUDA_CHECK(hipMalloc(&d_x, sizeof(float)*np2));
//  CUDA_CHECK(hipMalloc(&d_y, sizeof(float)*np2));


//  CUDA_CHECK(hipMemset(d_y, 0, sizeof(float)*np2));
  
  std::vector<float> rv(n);  // gpu result
  /*
  // prescan
  CUDA_CHECK(hipMemset(d_x, 0, sizeof(float)*np2));
  CUDA_CHECK(hipMemcpy(d_x, x.data(), sizeof(float)*n,
			hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_y, 0, sizeof(float)*np2));
  CUDA_CHECK(hipDeviceSynchronize());


  printf("calling prescan<<<%d,%d,%d>>> of %d elements.\n",
	 nblocks, blocksize, np2, n);

  begin = std::chrono::high_resolution_clock::now();
  prescan<<<nblocks, blocksize, np2*sizeof(float)>>>(d_x, d_y, n);

  hipDeviceSynchronize();
  end = std::chrono::high_resolution_clock::now();
  double duration_gpu = std::chrono::duration<double>(end - begin).count();

  hipMemcpy(rv.data(), d_y, sizeof(float)*n,
			hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  checkResult(x, rv, rv_cpu, duration_gpu, duration_cpu);
  */
  
  // segscan
  std::vector<float> flags(n,0.0);
  
  for (int i = 1; i < n; i*=2) {
      flags[i-1] = 1.0;
  }

  rv_cpu = cpu_segscan(x, flags);
  
  float* d_f;
  CUDA_CHECK(hipMalloc(&d_f, sizeof(float)*np2));
  CUDA_CHECK(hipMemset(d_f, 0, sizeof(float)*np2));
  CUDA_CHECK(hipMemcpy(d_f, flags.data(), sizeof(float)*n,
			hipMemcpyHostToDevice));

//  std::vector<float> flags_scan(n);
//  float* d_fs;
//  CUDA_CHECK(hipMalloc(&d_fs, sizeof(float)*np2));

//  std::vector<float> sums(nblocks), sumscan(nblocks), sums_flags(nblocks), sums_fs(nblocks);
//  float* d_sums, *d_sumscan, *d_sums_flags, *d_sums_fs;
//  CUDA_CHECK(hipMalloc(&d_sums, sizeof(float)*nblocks));
//  CUDA_CHECK(hipMalloc(&d_sumscan, sizeof(float)*nblocks));
//  CUDA_CHECK(hipMalloc(&d_sums_flags, sizeof(float)*nblocks));
//  CUDA_CHECK(hipMalloc(&d_sums_fs, sizeof(float)*nblocks));

  hipMemset(d_x, 0.0, sizeof(float)*np2);
  hipMemcpy(d_x, x.data(), sizeof(float)*n, hipMemcpyHostToDevice);
//  hipMemset(d_y, 0.0, sizeof(float)*np2);
  
  hipDeviceSynchronize();

  printf("calling segscan<<<%d,%d>>> of %d elements.\n",
	 nblocks, blocksize, n);
  begin = std::chrono::high_resolution_clock::now();
//    for (size_t i = 0; i < nblocks; i += block_n) {
//      size_t data_offset = i * block_n;
//      segscan<<<nblocks, blocksize, 3*block_n*sizeof(float)>>>(d_x + data_offset, d_y + data_offset, d_f + data_offset, d_fs, d_sums, d_sums_flags, block_n);
//      segscan<<<1, (nblocks+2-1)/2, 3*nblocks*sizeof(float)>>>(d_sums, d_sumscan, d_sums_flags, d_sums_fs, nullptr, nullptr, nblocks);
//      add_offsets<<<(np2+blocksize-1)/blocksize, blocksize>>>(d_y + data_offset, d_fs, d_sumscan, np2);
//    }

  d_y = run_segscan(d_x, d_f, n);

  hipDeviceSynchronize();
  end = std::chrono::high_resolution_clock::now();
  float duration_gpu = std::chrono::duration<double>(end - begin).count();

  hipMemcpy(rv.data(), d_y, sizeof(float)*n, hipMemcpyDeviceToHost);
//  hipMemcpy(flags_scan.data(), d_fs, sizeof(float)*n, hipMemcpyDeviceToHost);
//  hipMemcpy(sums.data(), d_sums, sizeof(float)*nblocks, hipMemcpyDeviceToHost);
//  hipMemcpy(sumscan.data(), d_sumscan, sizeof(float)*nblocks, hipMemcpyDeviceToHost);
//  hipMemcpy(sums_flags.data(), d_sums_flags, sizeof(float)*nblocks, hipMemcpyDeviceToHost);
//  hipMemcpy(sums_fs.data(), d_sums_fs, sizeof(float)*nblocks, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
//  printf("Flags are: ");
//  print(flags);
//  printf("\n");
//  printf("Flags scan are: ");
//  print(flags_scan);
//  printf("\n");
//  printf("Sums are: ");
//  print(sums);
//  printf("\n");
//  printf("Sums scan are: ");
//  print(sumscan);
//  printf("\n");
//  printf("Sums flags are: ");
//  print(sums_flags);
//  printf("\n");
//  printf("Sums fs are: ");
//  print(sums_fs);
//  printf("\n");
  checkResult(x, rv, rv_cpu, duration_gpu, duration_cpu);
  
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_f);

  return 0;
}
